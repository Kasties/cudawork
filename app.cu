
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void matrixmulkernel(float* M, float* N, float* P, int width) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < width && col < width) {
        float Pvalue = 0;
        for (int k = 0; k < width; k++) {
            Pvalue += M[row * width + k] * N[k * width + col];
        }
        P[row * width + col] = Pvalue;
    }
}

int main() {
    int width = 3;
    float M[width][width], N[width][width], P[width][width];
    float *Md, *Nd, *Pd;
    int size = width * width * sizeof(float);

    for (int i = 0; i < width; i++) {
        for (int j = 0; j < width; j++) {
            M[i][j] = 2;
            N[i][j] = 4;
        }
    }

    hipMalloc((void**)&Md, size);
    hipMalloc((void**)&Nd, size);
    hipMalloc((void**)&Pd, size);

    hipMemcpy(Md, M, size, hipMemcpyHostToDevice);
    hipMemcpy(Nd, N, size, hipMemcpyHostToDevice);

    dim3 dimBlock(width, width);
    dim3 dimGrid(1, 1);

    matrixmulkernel<<<dimGrid, dimBlock>>>(Md, Nd, Pd, width);

    hipMemcpy(P, Pd, size, hipMemcpyDeviceToHost);

    for (int i = 0; i < width; i++) {
        for (int j = 0; j < width; j++) {
            printf("%f ", P[i][j]);
        }
        printf("\n");
    }

    hipFree(Md);
    hipFree(Nd);
    hipFree(Pd);

    return 0;
}